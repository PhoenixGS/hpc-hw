#include "hip/hip_runtime.h"
#include "spmm_opt.h"

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{
    __shared__ int shared_idx[32];
    __shared__ float shared_val[32];
    float ans = 0.0;
    int indx = blockIdx.x;
    int indy = blockIdx.y * 32 + threadIdx.x;
    int begin = ptr[indx], end = ptr[indx + 1];
    for (int now = begin; now < end; now += 32)
    {
        if (now + threadIdx.x < end)
        {
            shared_idx[threadIdx.x] = idx[now + threadIdx.x];
            shared_val[threadIdx.x] = val[now + threadIdx.x];
        }
        __syncthreads();
        for (int i = 0; i < 32 && now + i < end; i++)
        {
            ans += shared_val[i] * vin[shared_idx[i] * INFEATURE + indy];
        }
    }
    vout[indx * INFEATURE + indy] = ans;
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    int WARP_SIZE = 32;
    grid.x = num_v;
    grid.y = (feat_in + WARP_SIZE - 1) / WARP_SIZE;
    block.x = WARP_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    spmm_kernel_placeholder<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}
