#include "hip/hip_runtime.h"
#include "spmm_opt.h"

const int C = 2;

__global__ void spmm_kernel_placeholder(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE)
{
    __shared__ int shared_idx[32];
    __shared__ float shared_val[32];
    float ans[C];
    for (int i = 0; i < C; i++)
    {
        ans[i] = 0.0;
    }
    int indx = blockIdx.x;
    int indy = blockIdx.y * 32 * C + threadIdx.x;
    int begin = ptr[indx], end = ptr[indx + 1];
    for (int now = begin; now < end; now += 32)
    {
        if (now + threadIdx.x < end)
        {
            shared_idx[threadIdx.x] = idx[now + threadIdx.x];
            shared_val[threadIdx.x] = val[now + threadIdx.x];
        }
        __syncthreads();
        for (int i = 0; i < 32 && now + i < end; i++)
        {
            for (int j = 0; j < C; j++)
            {
                ans[j] += shared_val[i] * vin[shared_idx[i] * INFEATURE + indy + j * 32];
            }
        }
    }
    for (int j = 0; j < C; j++)
    {
        vout[indx * INFEATURE + indy + j * 32] = ans[j];
    }
}
void SpMMOpt::preprocess(float *vin, float *vout)
{
    int WARP_SIZE = 32;
    grid.x = num_v;
    grid.y = (feat_in + WARP_SIZE * C - 1) / WARP_SIZE / C;
    block.x = WARP_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    spmm_kernel_placeholder<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
}
